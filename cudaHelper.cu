#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>
#include <omp.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include "helper.h"

#define gpuErrchk(ans)                        \
    {                                         \
        gpuAssert((ans), __FILE__, __LINE__); \
    }
// this gpuErrchk macro was taken from this link: https://stackoverflow.com/questions/14038589/what-is-the-canonical-way-to-check-for-errors-using-the-cuda-runtime-api
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true)
{
    if (code != hipSuccess)
    {
        fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort)
            exit(code);
    }
}

__device__ void calculateMatch(int objectDimension, int pictureDimension, int* d_pictureColorsMatrix, int* d_objectSubColorsMatrix, int picrureRow, int pictureCol, double* res)
{
        for( int i = 0; i < objectDimension; i++)
        {
            for( int j = 0; j < objectDimension; j++)
            {
                int objectColor = d_objectSubColorsMatrix[i * objectDimension + j];
                int pictureColor = d_pictureColorsMatrix[(picrureRow + i) * pictureDimension + (pictureCol + j)];
                if (pictureColor != 0)
                    *(res) += (double)abs((pictureColor - objectColor)) / pictureColor;
            }
        }
}

/*
 * Kernel function for calculating the difference between the colors of the overlapping pixels of the Object and the Picture using the formula:abs((P - O) / P)
 * @param d_pictureColorsMatrix - the colors matrix of the Picture on the GPU
 * @param d_objectSubColorsMatrix - the sub colors matrix of the Object on the GPU
 * @param d_matchingValue - the matching value that will be returned to the host
 * @param d_objectDimension - the dimension of the Object
 * @param d_pictureDimension - the dimension of the Picture
 * @param d_upperLeftCorner - the index of the upper-left corner of the object in the picture
 */
__global__ void calculateMatching(int *d_pictureColorsMatrix, int *d_objectSubColorsMatrix, double *d_matchingThreshold, int *d_objectDimension, int *d_pictureDimension, int *d_upperLeftCorner)
{
    int globalThreadIndex = blockDim.x * blockIdx.x + threadIdx.x;

    if (globalThreadIndex < ((*d_pictureDimension) - (*d_objectDimension) + 1) * ((*d_pictureDimension) - (*d_objectDimension) + 1))
    {
        double res;
        int pictureRow = globalThreadIndex / ((*d_pictureDimension) - (*d_objectDimension) + 1);
        int pictureCol = globalThreadIndex % ((*d_pictureDimension) - (*d_objectDimension) + 1);
        if (pictureCol < 0 || pictureCol >= (*d_pictureDimension) - (*d_objectDimension) + 1 || pictureRow < 0 || pictureRow >= (*d_pictureDimension) - (*d_objectDimension) + 1)
            return;
        calculateMatch(*d_objectDimension, *d_pictureDimension, d_pictureColorsMatrix, d_objectSubColorsMatrix, pictureRow, pictureCol, &res);
        if (res / ((*d_objectDimension) * (*d_objectDimension)) < (*d_matchingThreshold))
            (*d_upperLeftCorner) = pictureRow * (*d_pictureDimension) + pictureCol;
    }
}

__host__ void calculateMatchingOnGPU(Picture *picture, Object *object, int *upperLeftCorner, double matchingThreshold)
{
    // Allocate and copy memory for the matchingThreshold on the GPU
    double *d_matchingThreshold;
    gpuErrchk(hipMalloc((void **)&d_matchingThreshold, sizeof(double)));
    gpuErrchk(hipMemcpy(d_matchingThreshold, &matchingThreshold, sizeof(double), hipMemcpyHostToDevice));

    // Allocate memory and copy for the upper left corner on the GPU
    int *d_upperLeftCorner;
    gpuErrchk(hipMalloc((void **)&d_upperLeftCorner, sizeof(int)));
    gpuErrchk(hipMemcpy(d_upperLeftCorner, upperLeftCorner, sizeof(int), hipMemcpyHostToDevice));

    // Allocate memory and copy for the picture colors matrix on the GPU
    int *d_pictureColorsMatrix;
    gpuErrchk(hipMalloc((void **)&d_pictureColorsMatrix, picture->dimension * picture->dimension * sizeof(int)));
    gpuErrchk(hipMemcpy(d_pictureColorsMatrix, picture->colorsMatrix, picture->dimension * picture->dimension * sizeof(int), hipMemcpyHostToDevice));

    // Allocate memory and copy for the picture dimension on the GPU
    int *d_pictureDimension;
    gpuErrchk(hipMalloc((void **)&d_pictureDimension, sizeof(int)));
    gpuErrchk(hipMemcpy(d_pictureDimension, &picture->dimension, sizeof(int), hipMemcpyHostToDevice));

    // Allocate memory and copy for the object dimension on the GPU
    int *d_objectDimension;
    gpuErrchk(hipMalloc((void **)&d_objectDimension, sizeof(int)));
    gpuErrchk(hipMemcpy(d_objectDimension, &object->dimension, sizeof(int), hipMemcpyHostToDevice));

    // Allocate memory and copy for the object sub colors matrix on the GPU
    int *d_objectSubColorsMatrix;
    gpuErrchk(hipMalloc((void **)&d_objectSubColorsMatrix, (object->dimension * object->dimension * sizeof(int))));
    gpuErrchk(hipMemcpy(d_objectSubColorsMatrix, object->subColorsMatrix, (object->dimension * object->dimension * sizeof(int)), hipMemcpyHostToDevice));

    int size = (picture->dimension - object->dimension + 1) * (picture->dimension - object->dimension + 1);
    int blocksPerGrid = (size + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;

    // call the kernel function
    calculateMatching<<<blocksPerGrid, THREADS_PER_BLOCK>>>(d_pictureColorsMatrix, d_objectSubColorsMatrix, d_matchingThreshold, d_objectDimension, d_pictureDimension, d_upperLeftCorner);

    // check if the kernel function was called successfully
    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipDeviceSynchronize());

    // copy the upper left corner row from the GPU to the host
    gpuErrchk(hipMemcpy(upperLeftCorner, d_upperLeftCorner, sizeof(int), hipMemcpyDeviceToHost));

    // free the memory on the GPU
    hipFree(d_matchingThreshold);
    hipFree(d_pictureColorsMatrix);
    hipFree(d_objectSubColorsMatrix);
    hipFree(d_pictureDimension);
    hipFree(d_objectDimension);
    hipFree(d_upperLeftCorner);
}
